#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "CudaSimHandler.cuh"

#include <stdio.h>
#include <iostream>

hipError_t simWithCuda(CellState* cellGrid, int sizeX, int sizeY, int steps);

__global__ void cellSimKernel(CellState* inGrid, CellState* outGrid, int sizeX, int sizeY)
{
	int x = blockIdx.x * CELLBLOCKSIZEX + threadIdx.x;
	int y = blockIdx.y * CELLBLOCKSIZEY + threadIdx.y;
	int i = x * sizeX + y;
	if (x == 0 || x == sizeY-1 || y == 0 || y == sizeX-1) {
		outGrid[i] = DEAD;
		return;
	}
 
	int alive = inGrid[i - 1];
	alive += inGrid[i + 1];
	alive += inGrid[i - sizeX];
	alive += inGrid[i - sizeX + 1];
	alive += inGrid[i - sizeX - 1];
	alive += inGrid[i + sizeX];
	alive += inGrid[i + sizeX + 1];
	alive += inGrid[i + sizeX - 1];

	if (inGrid[i] == DEAD) { 
		if (alive != ANIMATEDEAD) {
			outGrid[i] = DEAD;
			return;
		}
	}
	else {
		if (alive < ISOLATION || alive > SMOTHERING) {
			outGrid[i] = DEAD;
			return;
		}
	}
	outGrid[i] = ALIVE;
}

int SimStep(CellState* cellGrid, vec size, int steps) {
	hipError_t cudaStatus = simWithCuda(cellGrid, size.x, size.y, steps);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "simWithCuda failed!");
		return 1;
	}
	return 0;
}

hipError_t simWithCuda(CellState* cellGrid, int sizeX, int sizeY, int steps) {

	CellState* dev_inGrid;
	CellState* dev_outGrid;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	int size = sizeX * sizeY;

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&dev_inGrid, sizeof(CellState) * size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;
	}
	cudaStatus = hipMalloc((void**)&dev_outGrid, sizeof(CellState) * size);
	if (cudaStatus != hipSuccess) {
		hipFree(dev_inGrid);
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;
	}

	// Copy the cellGrid to the gpu
	cudaStatus = hipMemcpy(dev_inGrid, cellGrid, sizeof(CellState) * size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 blocks(sizeX / CELLBLOCKSIZEX, sizeY / CELLBLOCKSIZEY);
	dim3 threads(CELLBLOCKSIZEX, CELLBLOCKSIZEY);

	//Handle multiple possible simsteps
	for (size_t i = 0; i < steps; i++) {

		if (i != 0) {
			CellState* temp = dev_inGrid;
			dev_inGrid = dev_outGrid;
			dev_outGrid = temp;
		}

		// Launch a kernel on the GPU
		cellSimKernel << <blocks, threads >> > (dev_inGrid, dev_outGrid, sizeX, sizeY);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cellSimKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(cellGrid, dev_outGrid, sizeof(CellState) * size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	//Free the gpu allocations
	hipFree(dev_inGrid);
	hipFree(dev_outGrid);

	return cudaStatus;
}

//Called at the end of the program
int CudaCleanup() {
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	if (hipDeviceReset() != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}
